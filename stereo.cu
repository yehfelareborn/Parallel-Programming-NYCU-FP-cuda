#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stereo.h"
#define BLOCK_SIZE 32

// 用來調整window size 比較不同window size花費的時間
int winSize     = 7;
int searchRange = 100;

// 方便進行block matching
int halfWinSize     = winSize /  2;
int halfSearchRange = searchRange / 2;

// GPU 的 Kernel

__global__ void Calculate(float *d_imgSrc, float *d_imgDst, float *d_disparity, int halfWinSize, int halfSearchRange)
{
    // 根據 CUDA 模型，算出當下 thread 對應的 x 與 y
    const int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = idx_y * 881 + idx_x;
    if(idx_y>2&&idx_y<402){
        if(idx_x>2&&idx_x<879){
        int cxDstMin = max(idx_x - halfSearchRange, halfWinSize); //left edge
        int cxDstMax = min(idx_x + halfSearchRange, 881-halfWinSize);//right edge

        float minSad        = FLT_MAX;
        float bestDisparity = 0;

        for (int cxDst = cxDstMin; cxDst < cxDstMax; ++cxDst) {
				float sad=0.f ;
                for(int y = -halfWinSize; y < halfWinSize+1; y++) {
                    for (int x = -halfWinSize; x < halfWinSize+1; x++) {
                    int idx_xWindow = idx_x+x;
                    int idx_yWindow = idx_y+y;
                    int idx_Window_Src =idx_yWindow * 881 + idx_xWindow;
                    int idx_Window_Dst =idx_yWindow * 881 + cxDst;
                    sad += abs(d_imgSrc[idx_Window_Src] - d_imgDst[idx_Window_Dst]);
                    }
                }
				if ( sad < minSad ) {
					minSad        = sad;
					bestDisparity = abs( cxDst - idx_x );
				}
        }
        d_disparity[idx]=bestDisparity;
        }
    }

}

void stereoMatch(const cv::Mat1f &imgSrc, const cv::Mat1f &imgDst, cv::Mat1f &disparity) {
    float* d_imgSrc; float* d_imgDst; float* d_disparity;
    hipMalloc(&d_imgSrc,  imgSrc.total() * sizeof(float));
    hipMalloc(&d_imgDst,  imgDst.total() * sizeof(float));
    disparity = cv::Mat1f::zeros( imgSrc.size() );
    hipMalloc(&d_disparity,  disparity.total() * sizeof(float));
    hipMemcpy(d_imgSrc, imgSrc.ptr<float>(0), imgSrc.total() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_imgDst, imgDst.ptr<float>(0), imgDst.total() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_disparity, disparity.ptr<float>(0), disparity.total() * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(881 / BLOCK_SIZE+1, 400 / BLOCK_SIZE+1);
    Calculate<<<numBlock, blockSize>>>(d_imgSrc, d_imgDst, d_disparity, halfWinSize, halfSearchRange);
    hipDeviceSynchronize();
    hipMemcpy(disparity.ptr(), d_disparity, disparity.total() * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_imgDst);
	hipFree(d_imgSrc);
    hipFree(d_disparity);
}
